#include "hip/hip_runtime.h"

#include <iostream>
#include <fstream>
#include <string>
#include <sstream>
#include <iostream>
#include <vector>
#include <iostream>
#include <fstream>
#include <algorithm>
#include <iomanip>
#include <cstring>
#include <chrono>
using namespace std::chrono;
using namespace std;
using std::string;
using std::cout;
using std::endl;


struct Edge
{
	int Source;
	int Destination;
	int Weight;
};


struct Graph {
	int V, E;
	struct Edge* edge;
};

struct Graph* createGraph(int V, int E)
{
	struct Graph* graph = new Graph;
	graph->V = V;
	graph->E = E;
	graph->edge = new Edge[E];
	return graph;
}

#define cudaSafeCall(call) {  \
  hipError_t err = call;       \
  if(hipSuccess != err){     \
    fprintf(stderr, "cudaSafeCall: %s(%i) : %s.\n", __FILE__, __LINE__, hipGetErrorString(err));   \
    exit(EXIT_FAILURE);       \
}}

#define cudaCheckErr(errorMessage) {    \
  hipError_t err = hipGetLastError(); \
  if(hipSuccess != err){               \
    fprintf(stderr, "hipError_t: %s(%i) : %s : %s.\n", __FILE__, __LINE__, errorMessage, hipGetErrorString(err)); \
    exit(EXIT_FAILURE);                 \
}}


void printArr(int dist[], int n)
{
	printf("10 Vertex Distance from Source\n");
	for (int i = 0; i < 10 && i < n; ++i)
		printf("%d \t\t %d\n", i, dist[i]);
}


void Print(int distance[], int count)
{
	ofstream myfile;
	myfile.open("results_cuda.txt");

	for (int i = 0; i < count; ++i)
		myfile << distance[i] << "\n";


	myfile.close();
}

__global__ void relax(struct Edge* edges, int E, int* dist) {
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if (tid < E) {
		int u = edges[tid].Source;
		int v = edges[tid].Destination;
		int weight = edges[tid].Weight;
		if (dist[u] != INT_MAX && dist[u] + weight < dist[v])
			dist[v] = dist[u] + weight;
	}
}


__global__ void checkNegCycle(struct Edge* edges, int E, int* dist) {
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if (tid < E) {
		int u = edges[tid].Source;
		int v = edges[tid].Destination;
		int weight = edges[tid].Weight;
		if (dist[u] != INT_MAX
			&& dist[u] + weight < dist[v])
			printf("Graph contains negative weight cycle");
		return;
	}
}


void BellmanFordParallel(struct Graph* graph, int* dist, int src)
{
	int V = graph->V;
	int E = graph->E;

	for (int i = 0; i < V; i++)
		dist[i] = INT_MAX;
	dist[src] = 0.0;

	int* d_dist;
	hipMalloc(&d_dist, V * sizeof(int));
	hipMemcpy(d_dist, dist, V * sizeof(int), hipMemcpyHostToDevice);


	struct Edge* edges = (struct Edge*)malloc(sizeof(struct Edge) * E);
	for (int i = 0; i < E; i++) {
		edges[i] = graph->edge[i];
	}
	struct Edge* d_edges;
	hipMalloc((void**)&d_edges, sizeof(struct Edge) * E);
	hipMemcpy(d_edges, edges, sizeof(struct Edge) * E, hipMemcpyHostToDevice);

	for (int i = 1; i <= V - 1; i++) {

		relax << <1 + (E - 1) / 1024, 1024 >> > (d_edges, E, d_dist);
	}

	checkNegCycle << <1 + (E - 1) / 1024, 1024 >> > (d_edges, E, d_dist);

	cudaSafeCall(hipDeviceSynchronize());
	cudaCheckErr("kernel error");

	hipMemcpy(dist, d_dist, V * sizeof(int), hipMemcpyDeviceToHost);

	Print(dist, V);

	free(edges);
	hipFree(d_edges);
	hipFree(d_dist);

	return;
}


void BellmanFordSerial(struct Graph* graph, int* dist, int src)
{
	int V = graph->V;
	int E = graph->E;

	for (int i = 0; i < V; i++) {
		dist[i] = INT_MAX;
	}

	dist[src] = 0;

	for (int i = 1; i <= V - 1; i++) {

		for (int j = 0; j < E; j++) {
			int u = graph->edge[j].Source;
			int v = graph->edge[j].Destination;
			int weight = graph->edge[j].Weight;
			if (dist[u] != INT_MAX
				&& dist[u] + weight < dist[v])
				dist[v] = dist[u] + weight;
		}

	}

	for (int i = 0; i < E; i++) {
		int u = graph->edge[i].Source;
		int v = graph->edge[i].Destination;
		int weight = graph->edge[i].Weight;
		if (dist[u] != INT_MAX
			&& dist[u] + weight < dist[v]) {
			printf("Graph contains negative weight cycle");
			return;
		}
	}

	Print(dist, V);

	return;
}

void checkSum(int distS[], int distP[], int size) {
	int count1 = 0;
	int count2 = 0;
	for (int i = 0; i < size; i++) {
		count1 += distS[i];
		count2 += distP[i];
	}
	printf("Sum of distances for Serial: %d.\nSum of distances for Parallel: %d.\n", count1, count2);
}

struct Graph* loadData(std::string filename) {

	int N;
	int edgesCount;
	std::ifstream inputf(filename, std::ifstream::in);
	int counter = 0;
	inputf >> N >> edgesCount;

	struct Graph* graph = createGraph(N, edgesCount);

	for (int i = 0; i < edgesCount; i++)
	{
		inputf >> graph->edge[i].Source >> graph->edge[i].Destination >> graph->edge[i].Weight;
	}

	return graph;
}


int main(int argc, char** argv)
{

	if (argc <= 1) {
		cout << "INPUT FILE WAS NOT FOUND!";
		return;
	}
	if (argc <= 2) {
		cout << "NUMBER OF THREADS WAS NOT FOUND!";
		return;
	}

	string filename = argv[1];
	struct Graph* graph = loadData(filename);
	cout << graph->V;


	int* distP = (int*)malloc(sizeof(int) * graph->V);
	int* distS = (int*)malloc(sizeof(int) * graph->V);
	auto start = high_resolution_clock::now();
	//BellmanFordSerial(graph, distS, 0);

	auto stop = high_resolution_clock::now();
	auto duration = duration_cast<microseconds>(stop - start);
	cout << "Bellman serial took " << duration.count() << " ms\n";
	cout << "---------------------------------\n";


	start = high_resolution_clock::now();
	BellmanFordParallel(graph, distP, 0);
	stop = high_resolution_clock::now();
	 duration = duration_cast<microseconds>(stop - start);
	 cout << "Bellman parallel took "<< duration.count() << " ms\n";

	 cout << "---------------------------------\n";
	checkSum(distS, distP, graph->V);


	free(distP);
	free(distS);
	return 0;
}

